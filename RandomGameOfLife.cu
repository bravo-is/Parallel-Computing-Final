#include "hip/hip_runtime.h"
/*
MODIFIED FROM CUDA BY EXAMPLE CH.7
Code Sustantially Modified into Conway's Game of Life by
Israel Bravo, Smit Patel, Prathamesh Bramhankar
for UC-Parallel Computing-Fall Semester-2019
*/


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"
#include <stdlib.h>
#include <time.h>

#define DIM 1024

// these exist on the GPU side
texture<float,2>  texIn;
texture<float,2>  texOut;

__global__ void GOL_kernel( float *dst, bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b, tl, tr, bl, br, neighbors;
    if (dstOut) {
      t = tex2D(texIn,x,y-1);//top
      l = tex2D(texIn,x-1,y);//left
      c = tex2D(texIn,x,y);//center
      r = tex2D(texIn,x+1,y);//right
      b = tex2D(texIn,x,y+1);//bottom
      tl = tex2D(texIn,x-1,y-1);//top-left
      tr = tex2D(texIn,x+1,y-1);//top-right
      bl = tex2D(texIn,x-1,y+1);//bottom-left
      br = tex2D(texIn,x+1,y+1);//bottom-right
    }else{
      t = tex2D(texOut,x,y-1);//top
      l = tex2D(texOut,x-1,y);//left
      c = tex2D(texOut,x,y);//center
      r = tex2D(texOut,x+1,y);//right
      b = tex2D(texOut,x,y+1);//bottom
      tl = tex2D(texOut,x-1,y-1);//top-left
      tr = tex2D(texOut,x+1,y-1);//top-right
      bl = tex2D(texOut,x-1,y+1);//bottom-left
      br = tex2D(texOut,x+1,y+1);//bottom-right
    }
    neighbors = t+l+r+b+tl+tr+bl+br;
    //Game of Life Rules
    if ( c == 1.0f && neighbors < 2.0f ){
      dst[offset] = 0.0f;
    }
    else if ( c == 1.0f && (neighbors == 2.0f || neighbors == 3.0f) ){
      dst[offset] = 1.0f;
    }
    else if ( c == 1.0f && neighbors > 3.0f ){
      dst[offset] = 0.0f;
    }
    else if ( c == 0.0f && neighbors == 3.0f ){
      dst[offset] = 1.0f;
    }
    else {
      dst[offset] = c;
    }
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    CPUAnimBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

void anim_gpu( DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    // we maintain this so that cylce speed can be controlled by timesteps or FPS
    volatile bool dstOut = true;
    for (int i=0; i<2; i++) {
        float *out;
        if (dstOut) {
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
        }
        GOL_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );

    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    CPUAnimBitmap bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    time_t t;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    HANDLE_ERROR( hipBindTexture2D( NULL, texIn,
                                   data.dev_inSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texOut,
                                   data.dev_outSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    /* Intializes random number generator */
    srand((unsigned) time(&t));

    // randomly populate the board
    float *cellState = (float*)malloc( imageSize );
    for (int i=0; i<DIM*DIM; i++) {
      if ( rand() % 2 == 0 ){
        cellState[i] = 0.0f;
      }else{
        cellState[i] = 1.0f;
      }
    }

    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, cellState,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( cellState );

    bitmap.anim_and_exit( (void (*)(void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}
